#include "hip/hip_runtime.h"
#include <mpi.h>

#include <cmath>
#include <cstdio>

#include "layer.h"
#include "model.h"

void print_device_pointer(float* d_ptr, size_t N) {
  float* h_ptr = (float*)malloc(N * sizeof(float));
  hipMemcpy(h_ptr, d_ptr, N * sizeof(float), hipMemcpyDeviceToHost);
  for (size_t i = 0; i < N; ++i) {
    printf("%lf ", h_ptr[i]);
  }
  printf("\n");
  free(h_ptr);
}

void print_device_pointer(int* d_ptr, size_t N) {
  int* h_ptr = (int*)malloc(N * sizeof(int));
  hipMemcpy(h_ptr, d_ptr, N * sizeof(int), hipMemcpyDeviceToHost);
  for (size_t i = 0; i < N; ++i) {
      std::cout << h_ptr[i] << " ";
  }
  std::cout << std::endl;
  free(h_ptr);
}

Parameter *attn_b[NUM_LAYER], *attn_w[NUM_LAYER];
Parameter *proj_b[NUM_LAYER], *proj_w[NUM_LAYER];
Parameter *ln_1_b[NUM_LAYER], *ln_1_g[NUM_LAYER];
Parameter *ln_2_b[NUM_LAYER], *ln_2_g[NUM_LAYER];
Parameter *mlp1_b[NUM_LAYER], *mlp1_w[NUM_LAYER];
Parameter *mlp2_b[NUM_LAYER], *mlp2_w[NUM_LAYER];
Parameter *ln_f_b, *ln_f_g;
Parameter *wpe, *wte;

/* Device pointers */
float *d_attn_b[NUM_LAYER], *d_attn_w[NUM_LAYER];
float *d_proj_b[NUM_LAYER], *d_proj_w[NUM_LAYER];
float *d_ln_1_b[NUM_LAYER], *d_ln_1_g[NUM_LAYER];
float *d_ln_2_b[NUM_LAYER], *d_ln_2_g[NUM_LAYER];
float *d_mlp1_b[NUM_LAYER], *d_mlp1_w[NUM_LAYER];
float *d_mlp2_b[NUM_LAYER], *d_mlp2_w[NUM_LAYER];
float *d_ln_f_b, *d_ln_f_g;
float *d_wpe, *d_wte;

void alloc_and_set_parameters(float *param) {
  size_t pos = 0;
  int order[] = {
      0, 1, 10, 11, 2, 3, 4, 5, 6, 7, 8, 9,
  };
  for (int i = 0; i < NUM_LAYER; i++) {
    attn_b[order[i]] = new Parameter({3 * HIDDEN_DIM}, param + pos);
    pos += OFFSET1;
    attn_w[order[i]] = new Parameter({HIDDEN_DIM, 3 * HIDDEN_DIM}, param + pos);
    pos += OFFSET2;
    proj_b[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    proj_w[order[i]] = new Parameter({HIDDEN_DIM, HIDDEN_DIM}, param + pos);
    pos += OFFSET4;
    ln_1_b[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    ln_1_g[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    ln_2_b[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    ln_2_g[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    mlp1_b[order[i]] = new Parameter({4 * HIDDEN_DIM}, param + pos);
    pos += OFFSET5;
    mlp1_w[order[i]] = new Parameter({HIDDEN_DIM, 4 * HIDDEN_DIM}, param + pos);
    pos += OFFSET6;
    mlp2_b[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    mlp2_w[order[i]] = new Parameter({4 * HIDDEN_DIM, HIDDEN_DIM}, param + pos);
    pos += OFFSET6;
  }
  ln_f_b = new Parameter({HIDDEN_DIM}, param + pos);
  pos += OFFSET3;
  ln_f_g = new Parameter({HIDDEN_DIM}, param + pos);
  pos += OFFSET3;
  wpe = new Parameter({MAX_SEQ_LEN, HIDDEN_DIM}, param + pos);
  pos += OFFSET7;
  wte = new Parameter({NUM_VOCAB, HIDDEN_DIM}, param + pos);
  pos += OFFSET8;

  /* Allocate device memory */
  for (int i = 0; i < NUM_LAYER; i++) {
    hipMalloc(&d_attn_b[order[i]], 3 * HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_attn_w[order[i]], HIDDEN_DIM * 3 * HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_proj_b[order[i]], HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_proj_w[order[i]], HIDDEN_DIM * HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_ln_1_b[order[i]], HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_ln_1_g[order[i]], HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_ln_2_b[order[i]], HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_ln_2_g[order[i]], HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_mlp1_b[order[i]], 4 * HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_mlp1_w[order[i]], HIDDEN_DIM * 4 * HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_mlp2_b[order[i]], HIDDEN_DIM * sizeof(float));
    hipMalloc(&d_mlp2_w[order[i]], 4 * HIDDEN_DIM * HIDDEN_DIM * sizeof(float));
  }
  hipMalloc(&d_ln_f_b, HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_ln_f_g, HIDDEN_DIM * sizeof(float));
  // hipMalloc(&d_wpe, MAX_SEQ_LEN * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_wpe, wpe->num_elem() * sizeof(float)); 
  // hipMalloc(&d_wte, NUM_VOCAB * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_wte, wte->num_elem() * sizeof(float));
  
  /* Copy data to device */
  for (int i = 0; i < NUM_LAYER; i++) {
    hipMemcpy(d_attn_b[order[i]], attn_b[order[i]]->buf, 3 * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_attn_w[order[i]], attn_w[order[i]]->buf, HIDDEN_DIM * 3 * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_proj_b[order[i]], proj_b[order[i]]->buf, HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_proj_w[order[i]], proj_w[order[i]]->buf, HIDDEN_DIM * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ln_1_b[order[i]], ln_1_b[order[i]]->buf, HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ln_1_g[order[i]], ln_1_g[order[i]]->buf, HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ln_2_b[order[i]], ln_2_b[order[i]]->buf, HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ln_2_g[order[i]], ln_2_g[order[i]]->buf, HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mlp1_b[order[i]], mlp1_b[order[i]]->buf, 4 * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mlp1_w[order[i]], mlp1_w[order[i]]->buf, HIDDEN_DIM * 4 * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mlp2_b[order[i]], mlp2_b[order[i]]->buf, HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mlp2_w[order[i]], mlp2_w[order[i]]->buf, 4 * HIDDEN_DIM * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  }
  hipMemcpy(d_ln_f_b, ln_f_b->buf, HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_ln_f_g, ln_f_g->buf, HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  // hipMemcpy(d_wpe, wpe->buf, MAX_SEQ_LEN * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_wpe, wpe->buf, wpe->num_elem() * sizeof(float), hipMemcpyHostToDevice); 
  // hipMemcpy(d_wte, wte->buf, NUM_VOCAB * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_wte, wte->buf, wte->num_elem() * sizeof(float), hipMemcpyHostToDevice);
}

void free_parameters() {
  for (int i = 0; i < NUM_LAYER; i++) {
    delete attn_b[i];
    delete attn_w[i];
    delete proj_b[i];
    delete proj_w[i];
    delete ln_1_b[i];
    delete ln_1_g[i];
    delete ln_2_b[i];
    delete ln_2_g[i];
    delete mlp1_b[i];
    delete mlp1_w[i];
    delete mlp2_b[i];
    delete mlp2_w[i];
  }
  delete ln_f_b;
  delete ln_f_g;
  delete wpe;
  delete wte;

  for (int i = 0; i < NUM_LAYER; i++) {
    hipFree(d_attn_b[i]);
    hipFree(d_attn_w[i]);
    hipFree(d_proj_b[i]);
    hipFree(d_proj_w[i]);
    hipFree(d_ln_1_b[i]);
    hipFree(d_ln_1_g[i]);
    hipFree(d_ln_2_b[i]);
    hipFree(d_ln_2_g[i]);
    hipFree(d_mlp1_b[i]);
    hipFree(d_mlp1_w[i]);
    hipFree(d_mlp2_b[i]);
    hipFree(d_mlp2_w[i]);
  }
  hipFree(d_ln_f_b);
  hipFree(d_ln_f_g);
  hipFree(d_wpe);
  hipFree(d_wte);
}

Activation *embd_a, *ffn_proj_a;
Activation *mha_qkv_proj_a, *mha_out_a, *mha_split_qkv_a, *mha_split_head_a,
    *mha_mask_a, *mha_merge_head_a, *mha_q_a, *mha_k_a, *mha_v_a,
    *mha_attn_out_a, *mha_concat_head_a;
Activation *attn_score_a, *k_transposed_a;
Activation *wte_transposed_a, *residual_a, *logit_a;
Activation *transformer_block_a;

float *d_embd_a, *d_ffn_proj_a;
float *d_mha_qkv_proj_a, *d_mha_out_a, *d_mha_split_qkv_a,
    *d_mha_split_head_a, *d_mha_mask_a, *d_mha_merge_head_a, *d_mha_q_a,
    *d_mha_k_a, *d_mha_v_a, *d_mha_attn_out_a, *d_mha_concat_head_a;
float *d_attn_score_a, *d_k_transposed_a;
float *d_wte_transposed_a, *d_residual_a, *d_logit_a;
float *d_transformer_block_a;

void alloc_activations(size_t prompt_size) {
  embd_a = new Activation({prompt_size, HIDDEN_DIM});

  ffn_proj_a = new Activation({prompt_size, 4 * HIDDEN_DIM});

  mha_qkv_proj_a = new Activation({prompt_size, 3 * HIDDEN_DIM});
  mha_out_a = new Activation({prompt_size, HIDDEN_DIM});
  mha_split_qkv_a = new Activation({3, prompt_size, HIDDEN_DIM});
  mha_split_head_a =
      new Activation({3, NUM_HEAD, prompt_size, HIDDEN_DIM / NUM_HEAD});
  mha_mask_a = new Activation({prompt_size, prompt_size});
  mha_merge_head_a =
      new Activation({NUM_HEAD, prompt_size, HIDDEN_DIM / NUM_HEAD});
  mha_q_a = new Activation({prompt_size, HIDDEN_DIM / NUM_HEAD});
  mha_k_a = new Activation({prompt_size, HIDDEN_DIM / NUM_HEAD});
  mha_v_a = new Activation({prompt_size, HIDDEN_DIM / NUM_HEAD});
  mha_attn_out_a = new Activation({prompt_size, HIDDEN_DIM / NUM_HEAD});
  mha_concat_head_a = new Activation({prompt_size, HIDDEN_DIM});

  attn_score_a = new Activation({prompt_size, prompt_size});
  k_transposed_a = new Activation({HIDDEN_DIM / NUM_HEAD, prompt_size});

  wte_transposed_a = new Activation({HIDDEN_DIM, NUM_VOCAB});

  residual_a = new Activation({prompt_size, HIDDEN_DIM});
  logit_a = new Activation({prompt_size, NUM_VOCAB});
  transformer_block_a = new Activation({prompt_size, HIDDEN_DIM});

  /* Allocate device memory */
  // hipMalloc(&d_embd_a, prompt_size * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_embd_a, embd_a->num_elem() * sizeof(float)); 
  hipMalloc(&d_ffn_proj_a, prompt_size * 4 * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_mha_qkv_proj_a, prompt_size * 3 * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_mha_out_a, prompt_size * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_mha_split_qkv_a, 3 * prompt_size * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_mha_split_head_a, 3 * NUM_HEAD * prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_mask_a, prompt_size * prompt_size * sizeof(float));
  hipMalloc(&d_mha_merge_head_a, NUM_HEAD * prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_q_a, prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_k_a, prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_v_a, prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_attn_out_a, prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_concat_head_a, prompt_size * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_attn_score_a, prompt_size * prompt_size * sizeof(float));
  hipMalloc(&d_k_transposed_a, HIDDEN_DIM / NUM_HEAD * prompt_size * sizeof(float));
  hipMalloc(&d_wte_transposed_a, HIDDEN_DIM * NUM_VOCAB * sizeof(float));
  hipMalloc(&d_residual_a, prompt_size * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_logit_a, prompt_size * NUM_VOCAB * sizeof(float));
  hipMalloc(&d_transformer_block_a, prompt_size * HIDDEN_DIM * sizeof(float));

  /* Copy data to device */
  // hipMemcpy(d_embd_a, embd_a->buf, prompt_size * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_embd_a, embd_a->buf, embd_a->num_elem() * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_ffn_proj_a, ffn_proj_a->buf, prompt_size * 4 * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mha_qkv_proj_a, mha_qkv_proj_a->buf, prompt_size * 3 * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mha_out_a, mha_out_a->buf, prompt_size * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mha_split_qkv_a, mha_split_qkv_a->buf, 3 * prompt_size * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mha_split_head_a, mha_split_head_a->buf, 3 * NUM_HEAD * prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mha_mask_a, mha_mask_a->buf, prompt_size * prompt_size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mha_merge_head_a, mha_merge_head_a->buf, NUM_HEAD * prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mha_q_a, mha_q_a->buf, prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mha_k_a, mha_k_a->buf, prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mha_v_a, mha_v_a->buf, prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mha_attn_out_a, mha_attn_out_a->buf, prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mha_concat_head_a, mha_concat_head_a->buf, prompt_size * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_attn_score_a, attn_score_a->buf, prompt_size * prompt_size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_k_transposed_a, k_transposed_a->buf, HIDDEN_DIM / NUM_HEAD * prompt_size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_wte_transposed_a, wte_transposed_a->buf, HIDDEN_DIM * NUM_VOCAB * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_residual_a, residual_a->buf, prompt_size * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_logit_a, logit_a->buf, prompt_size * NUM_VOCAB * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_transformer_block_a, transformer_block_a->buf, prompt_size * HIDDEN_DIM * sizeof(float), hipMemcpyHostToDevice);
}

void free_activations() {
  delete embd_a;
  delete ffn_proj_a;
  delete mha_qkv_proj_a;
  delete mha_out_a;
  delete mha_split_qkv_a;
  delete mha_split_head_a;
  delete mha_mask_a;
  delete mha_merge_head_a;
  delete mha_q_a;
  delete mha_k_a;
  delete mha_v_a;
  delete mha_attn_out_a;
  delete mha_concat_head_a;
  delete attn_score_a;
  delete k_transposed_a;
  delete wte_transposed_a;
  delete residual_a;
  delete logit_a;
  delete transformer_block_a;

  // hipFree(d_embd_a);
  // hipFree(d_ffn_proj_a);
  // hipFree(d_mha_qkv_proj_a);
  // hipFree(d_mha_out_a);
  // hipFree(d_mha_split_qkv_a);
  // hipFree(d_mha_split_head_a);
  // hipFree(d_mha_mask_a);
  // hipFree(d_mha_merge_head_a);
  // hipFree(d_mha_q_a);
  // hipFree(d_mha_k_a);
  // hipFree(d_mha_v_a);
  // hipFree(d_mha_attn_out_a);
  // hipFree(d_mha_concat_head_a);
  // hipFree(d_attn_score_a);
  // hipFree(d_k_transposed_a);
  // hipFree(d_wte_transposed_a);
  // hipFree(d_residual_a);
  // hipFree(d_logit_a);
  // hipFree(d_transformer_block_a);
}

/* (Position-wise) Feed-Forward Network
 * @param [in1]     in: [seq_len, HIDDEN_DIM]
 * @param [in2] mlp1_w: [HIDDEN_DIM, 4*HIDDEN_DIM]
 * @param [in3] mlp1_b: [4*HIDDEN_DIM]
 * @param [in4] mlp2_w: [4*HIDDEN_DIM, HIDDEN_DIM]
 * @param [in5] mlp2_b: [HIDDEN_DIM]
 * @param [out]    out: [seq_len, HIDDEN_DIM]
 */
// void ffn(Activation *in, Parameter *mlp1_w, Parameter *mlp1_b,
//          Parameter *mlp2_w, Parameter *mlp2_b, Activation *out) {
//   /* Projection Up:
//     [seq_len, HIDDEN_DIM] -> [seq_len, 4*HIDDEN_DIM] */
//   linear(in, mlp1_w, mlp1_b, ffn_proj_a);

//   /* GELU */
//   gelu(ffn_proj_a);

//   /* Projection Down:
//     [seq_len, 4*HIDDEN_DIM] -> [seq_len, HIDDEN_DIM] */
//   linear(ffn_proj_a, mlp2_w, mlp2_b, out);
// }
void ffn(float *d_in, float *d_mlp1_w, float *d_mlp1_b,
         float *d_mlp2_w, float *d_mlp2_b, float *d_out, size_t seq_len) {
    /* Projection Up: [seq_len, HIDDEN_DIM] -> [seq_len, 4*HIDDEN_DIM] */
    linear(d_in, d_mlp1_w, d_mlp1_b, d_ffn_proj_a, seq_len, HIDDEN_DIM, 4 * HIDDEN_DIM);

    /* GELU */
    gelu(d_ffn_proj_a, seq_len * 4 * HIDDEN_DIM);

    /* Projection Down: [seq_len, 4*HIDDEN_DIM] -> [seq_len, HIDDEN_DIM] */
    linear(d_ffn_proj_a, d_mlp2_w, d_mlp2_b, d_out, seq_len, 4 * HIDDEN_DIM, HIDDEN_DIM);
}

/* Attention
 * @param [in1]    q: [seq_len, HIDDEN_DIM/NUM_HEAD]
 * @param [in2]    k: [seq_len, HIDDEN_DIM/NUM_HEAD]
 * @param [in3]    v: [seq_len, HIDDEN_DIM/NUM_HEAD]
 * @param [in4] mask: [seq_len, HIDDEN_DIM/NUM_HEAD]
 * @param [out]  out: [seq_len, HIDDEN_DIM/NUM_HEAD]
 */
// void attention(Activation *q, Activation *k, Activation *v, Activation *mask,
//                Activation *out) {
//   /* Get Attention score by q @ k */
//   transpose(k, k_transposed_a);
//   matmul(q, k_transposed_a, attn_score_a);

//   /* Scaling */
//   scaling(attn_score_a, (1.0 / sqrt(k->shape[1])));

//   /* Masking */
//   add(attn_score_a, mask);

//   /* Softmax */
//   softmax(attn_score_a);

//   /* Attention score @ v */
//   matmul(attn_score_a, v, out);
// }
void attention(float *d_q, float *d_k, float *d_v, float *d_mask, float *d_out, size_t seq_len, size_t head_dim) {
    /* Get Attention score by q @ k */
    transpose(d_k, d_k_transposed_a, seq_len, head_dim);
    matmul(d_q, d_k_transposed_a, d_attn_score_a, seq_len, head_dim, seq_len);

    /* Scaling */
    scaling(d_attn_score_a, 1.0 / sqrt(head_dim), seq_len * seq_len);

    /* Masking */
    add(d_attn_score_a, d_mask, seq_len * seq_len);

    /* Softmax */
    // TODO handle HIDDEN_DIM
    softmax(d_attn_score_a, seq_len, HIDDEN_DIM);

    /* Attention score @ v */
    matmul(d_attn_score_a, d_v, d_out, seq_len, seq_len, head_dim);
}

/* (Masked) Multi-Head Self Attention
 * @param [in1]     in: [seq_len, HIDDEN_DIM]
 * @param [in2] attn_b: [3*HIDDEN_DIM]
 * @param [in3] attn_w: [HIDDEN_DIM, 3*HIDDEN_DIM]
 * @param [in4] proj_b: [HIDDEN_DIM]
 * @param [in5] proj_w: [HIDDEN_DIM, HIDDEN_DIM]
 * @param [out]    out: [seq_len, HIDDEN_DIM]
 */
// void mha(Activation *in, Parameter *attn_b, Parameter *attn_w,
//          Parameter *proj_b, Parameter *proj_w, Activation *out) {
//   /* QKV projection:
//     [seq_len, HIDDEN_DIM] ->
//     [seq_len, 3*HIDDEN_DIM] */
//   linear(in, attn_w, attn_b, mha_qkv_proj_a);

//   /* Split into Q, K, V:
//     [seq_len, 3*HIDDEN_DIM] ->
//     [3, seq_len, HIDDEN_DIM] */
//   split_qkv(mha_qkv_proj_a, mha_split_qkv_a);

//   /* Split into multiple heads:
//     [3, seq_len, HIDDEN_DIM] ->
//     [3, NUM_HEAD, seq_len, HIDDEN_DIM/NUM_HEAD] */
//   split_head(mha_split_qkv_a, NUM_HEAD, mha_split_head_a);

//   /* Generate mask to hide future inputs */
//   generate_mask(mha_mask_a);

//   /* Perform Attention over each head:
//     [NUM_HEAD, 3, seq_len, HIDDEN_DIM/NUM_HEAD] ->
//     [NUM_HEAD, seq_len, HIDDEN_DIM/NUM_HEAD] */
//   for (size_t idx = 0; idx < NUM_HEAD; idx++) {
//     /* Extract Q, K, V from qkv_head */
//     extract_qkv(mha_split_head_a, idx, NUM_HEAD, mha_q_a, mha_k_a, mha_v_a);

//     /* Attention */
//     attention(mha_q_a, mha_k_a, mha_v_a, mha_mask_a, mha_attn_out_a);

//     /* Merge each head's attn output
//       [seq_len, HIDDEN_DIM/NUM_HEAD] ->
//       [NUM_HEAD, seq_len, HIDDEN_DIM/NUM_HEAD] */
//     merge_head(mha_attn_out_a, idx, NUM_HEAD, mha_merge_head_a);
//   }

//   /* Concat each heads:
//     [NUM_HEAD, seq_len, HIDDEN_DIM/NUM_HEAD] ->
//     [seq_len, HIDDEN_DIM] */
//   concat_head(mha_merge_head_a, mha_concat_head_a);

//   /* OUT projection:
//     [seq_len, HIDDEN_DIM] -> [seq_len, HIDDEN_DIM] */
//   linear(mha_concat_head_a, proj_w, proj_b, out);
// }
void mha(float *d_in, float *d_attn_b, float *d_attn_w,
         float *d_proj_b, float *d_proj_w, float *d_out, size_t seq_len) {
    /* QKV projection: [seq_len, HIDDEN_DIM] -> [seq_len, 3*HIDDEN_DIM] */
    linear(d_in, d_attn_w, d_attn_b, d_mha_qkv_proj_a, seq_len, HIDDEN_DIM, 3 * HIDDEN_DIM);

    /* Split into Q, K, V: [seq_len, 3*HIDDEN_DIM] -> [3, seq_len, HIDDEN_DIM] */
    split_qkv(d_mha_qkv_proj_a, d_mha_split_qkv_a, seq_len, HIDDEN_DIM);

    /* Split into multiple heads: [3, seq_len, HIDDEN_DIM] -> [3, NUM_HEAD, seq_len, HIDDEN_DIM/NUM_HEAD] */
    split_head(d_mha_split_qkv_a, d_mha_split_head_a, NUM_HEAD, seq_len, HIDDEN_DIM);

    /* Generate mask to hide future inputs */
    generate_mask(d_mha_mask_a, seq_len);

    /* Perform Attention over each head: [NUM_HEAD, 3, seq_len, HIDDEN_DIM/NUM_HEAD] -> [NUM_HEAD, seq_len, HIDDEN_DIM/NUM_HEAD] */
    for (size_t idx = 0; idx < NUM_HEAD; idx++) {
        /* Extract Q, K, V from qkv_head */
        extract_qkv(d_mha_split_head_a, d_mha_q_a, d_mha_k_a, d_mha_v_a, idx, NUM_HEAD, seq_len, HIDDEN_DIM / NUM_HEAD);

        /* Attention */
        attention(d_mha_q_a, d_mha_k_a, d_mha_v_a, d_mha_mask_a, d_mha_attn_out_a, seq_len, HIDDEN_DIM / NUM_HEAD);

        /* Merge each head's attn output [seq_len, HIDDEN_DIM/NUM_HEAD] -> [NUM_HEAD, seq_len, HIDDEN_DIM/NUM_HEAD] */
        merge_head(d_mha_attn_out_a, d_mha_merge_head_a, idx, seq_len, HIDDEN_DIM / NUM_HEAD);
    }

    /* Concat each heads: [NUM_HEAD, seq_len, HIDDEN_DIM/NUM_HEAD] -> [seq_len, HIDDEN_DIM] */
    concat_head(d_mha_merge_head_a, d_mha_concat_head_a, NUM_HEAD, seq_len, HIDDEN_DIM / NUM_HEAD);

    /* OUT projection: [seq_len, HIDDEN_DIM] -> [seq_len, HIDDEN_DIM] */
    linear(d_mha_concat_head_a, d_proj_w, d_proj_b, d_out, seq_len, HIDDEN_DIM, HIDDEN_DIM);
}


/* Transformer Block
 * @param [in1]      in: [seq_len, HIDDEN_DIM]
 * @param [in2]  attn_b: [3*HIDDEN_DIM]
 * @param [in3]  attn_w: [HIDDEN_DIM, 3*HIDDEN_DIM]
 * @param [in4]  proj_b: [HIDDEN_DIM]
 * @param [in5]  proj_w: [HIDDEN_DIM, HIDDEN_DIM]
 * @param [in6]  ln_1_b: [HIDDEN_DIM]
 * @param [in7]  ln_1_g: [HIDDEN_DIM]
 * @param [in8]  ln_2_b: [HIDDEN_DIM]
 * @param [in9]  ln_2_g: [HIDDEN_DIM]
 * @param [in10] mlp1_b: [4*HIDDEN_DIM]
 * @param [in11] mlp1_w: [HIDDEN_DIM, 4*HIDDEN_DIM]
 * @param [in12] mlp2_b: [HIDDEN_DIM]
 * @param [in13] mlp2_w: [4*HIDDEN_DIM, HIDDEN_DIM]
 * @param [out]     out: [seq_len, HIDDEN_DIM]
 */
// void transformer_block(Activation *in, Parameter *attn_b, Parameter *attn_w,
//                        Parameter *proj_b, Parameter *proj_w, Parameter *ln_1_b,
//                        Parameter *ln_1_g, Parameter *ln_2_b, Parameter *ln_2_g,
//                        Parameter *mlp1_b, Parameter *mlp1_w, Parameter *mlp2_b,
//                        Parameter *mlp2_w, Activation *out) {
//   /* Copy Residual */
//   copy(in, residual_a);

//   /* Layer Normalization */
//   layer_norm(in, ln_1_g, ln_1_b);

//   /* Masked Multi-Head Self-Attention */
//   mha(in, attn_b, attn_w, proj_b, proj_w, mha_out_a);

//   /* Add Residual */
//   add(mha_out_a, residual_a);

//   /* Copy Residual */
//   copy(mha_out_a, residual_a);

//   /* Layer Normalization */
//   layer_norm(mha_out_a, ln_2_g, ln_2_b);

//   /* Position-wise Feed-Forward Network */
//   ffn(mha_out_a, mlp1_w, mlp1_b, mlp2_w, mlp2_b, out);

//   /* Add Residual */
//   add(out, residual_a);
// }
void transformer_block(float *d_in, float *d_attn_b, float *d_attn_w,
                       float *d_proj_b, float *d_proj_w, float *d_ln_1_b,
                       float *d_ln_1_g, float *d_ln_2_b, float *d_ln_2_g,
                       float *d_mlp1_b, float *d_mlp1_w, float *d_mlp2_b,
                       float *d_mlp2_w, float *d_out, size_t seq_len) { 
    /* Copy Residual */
    copy(d_in, d_residual_a, seq_len * HIDDEN_DIM);

    /* Layer Normalization */
    layer_norm(d_in, d_ln_1_g, d_ln_1_b, seq_len, HIDDEN_DIM, 1e-5);

    /* Masked Multi-Head Self-Attention */
    mha(d_in, d_attn_b, d_attn_w, d_proj_b, d_proj_w, d_mha_out_a, seq_len);

    /* Add Residual */
    add(d_mha_out_a, d_residual_a, seq_len * HIDDEN_DIM);

    /* Copy Residual */
    copy(d_mha_out_a, d_residual_a, seq_len * HIDDEN_DIM);

    /* Layer Normalization */
    layer_norm(d_mha_out_a, d_ln_2_g, d_ln_2_b, seq_len, HIDDEN_DIM, 1e-5);

    /* Position-wise Feed-Forward Network */
    ffn(d_mha_out_a, d_mlp1_w, d_mlp1_b, d_mlp2_w, d_mlp2_b, d_out, seq_len);

    /* Add Residual */
    add(d_out, d_residual_a, seq_len * HIDDEN_DIM);
}

/* [Model Computation: Token Generation] */
// void generate_tokens(int *input, int *output, size_t n_prompt, size_t n_token) {
//   int mpi_rank;
//   MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
//   if (mpi_rank == 0) {
//     /* Outer loop: generate tokens for each prompt */
//     for (size_t p = 0; p < n_prompt; p++) {
//       int prompt_size = tokens_per_prompt;

//       /* Initialize input prompt */
//       vector<int> input_prompt(prompt_size);
//       memcpy(input_prompt.data(), input + p * prompt_size,
//              prompt_size * sizeof(int));

//       /* Inner loop: generate next token */
//       for (size_t t = 0; t < n_token; t++) {
//         /* Initialize activations */
//         alloc_activations(prompt_size);

//         /* Token + Positional Embedding */
//         token_pos_embedding(input_prompt, wte, wpe, embd_a);

//         /* Forward path of Transformer blocks */
//         for (size_t l = 0; l < NUM_LAYER; l++) {
//           transformer_block(embd_a, attn_b[l], attn_w[l], proj_b[l], proj_w[l],
//                             ln_1_b[l], ln_1_g[l], ln_2_b[l], ln_2_g[l],
//                             mlp1_b[l], mlp1_w[l], mlp2_b[l], mlp2_w[l],
//                             transformer_block_a);

//           /* Copy output to embd_a for next block */
//           copy(transformer_block_a, embd_a);
//         }

//         /* Final Layer Normalization */
//         layer_norm(embd_a, ln_f_g, ln_f_b);

//         /* Projection to vocab. dimension */
//         transpose(wte, wte_transposed_a);
//         matmul(embd_a, wte_transposed_a, logit_a);

//         /* Greedy sampling (only last timestep is considered) */
//         int next_token_id = top1_sampling(logit_a);

//         /* Update input prompt and prompt size */
//         input_prompt.push_back(next_token_id);
//         prompt_size += 1;

//         /* Store generated token to output */
//         output[p * n_token + t] = next_token_id;

//         /* Finalize activations for next token generation */
//         free_activations();
//       }
//     }
//   }
// }
void generate_tokens(int *input, int *output, size_t n_prompt, size_t n_token) {
  int mpi_rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  if (mpi_rank == 0) {
    /* Outer loop: generate tokens for each prompt */
    for (size_t p = 0; p < n_prompt; p++) {
      int prompt_size = tokens_per_prompt;

      /* Initialize input prompt */
      vector<int> input_prompt(prompt_size);
      memcpy(input_prompt.data(), input + p * prompt_size,
             prompt_size * sizeof(int));

      // Temporary 
      int *d_input_prompt; 
      hipMalloc(&d_input_prompt, prompt_size * sizeof(int));
      hipMemcpy(d_input_prompt, input_prompt.data(), prompt_size * sizeof(int), hipMemcpyHostToDevice);

      /* Inner loop: generate next token */
      for (size_t t = 0; t < n_token; t++) {
        /* Initialize activations */
        alloc_activations(prompt_size);

        printf("\n");
        printf("d_wte: ");
        print_device_pointer(d_wte, 10);
        printf("d_wpe: ");
        print_device_pointer(d_wpe, 10);
        printf("d_input_prompt: ");
        print_device_pointer(d_input_prompt, 10);
        printf("d_embd_a: ");
        print_device_pointer(d_embd_a, 10);

        /* Token + Positional Embedding */
        token_pos_embedding(input_prompt, wte, wpe, embd_a);
        token_pos_embedding(d_input_prompt, d_wte, d_wpe, d_embd_a, prompt_size, HIDDEN_DIM);

        // printf("d_wte: ");
        // print_device_pointer(d_wte, 10);
        // printf("d_wpe: ");
        // print_device_pointer(d_wpe, 10);
        printf("d_embd_a: ");
        print_device_pointer(d_embd_a, 10);

        printf("embd_a: ");
        for (int i = 0; i < 10; i++) 
          printf("%f ", embd_a->buf[i]);
        printf("\n");

        /* Forward path of Transformer blocks */
        for (size_t l = 0; l < NUM_LAYER; l++) {
          // transformer_block(embd_a, attn_b[l], attn_w[l], proj_b[l], proj_w[l],
          //                   ln_1_b[l], ln_1_g[l], ln_2_b[l], ln_2_g[l],
          //                   mlp1_b[l], mlp1_w[l], mlp2_b[l], mlp2_w[l],
          //                   transformer_block_a);
          transformer_block(d_embd_a, d_attn_b[l], d_attn_w[l], d_proj_b[l], d_proj_w[l],
                                      d_ln_1_b[l], d_ln_1_g[l], d_ln_2_b[l], d_ln_2_g[l],
                                      d_mlp1_b[l], d_mlp1_w[l], d_mlp2_b[l], d_mlp2_w[l],
                                      d_transformer_block_a, prompt_size);

          /* Copy output to embd_a for next block */
          // copy(transformer_block_a, embd_a);
          copy(d_transformer_block_a, d_embd_a, prompt_size * HIDDEN_DIM);
        }

        /* Final Layer Normalization */
        // layer_norm(embd_a, ln_f_g, ln_f_b);
        layer_norm(d_embd_a, d_ln_f_g, d_ln_f_b, embd_a->shape[0], embd_a->shape[1], 1e-5);

        /* Projection to vocab. dimension */
        // transpose(wte, wte_transposed_a);
        transpose(d_wte, d_wte_transposed_a, wte->shape[0], wte->shape[1]);
        // matmul(embd_a, wte_transposed_a, logit_a);
        matmul(d_embd_a, d_wte_transposed_a, d_logit_a, embd_a->shape[0], wte->shape[0], wte->shape[1]);

        // printf("d_embd_a: ");
        // print_device_pointer(d_embd_a, 10);

        // TODO 
        hipMemcpy(logit_a->buf, d_logit_a, logit_a->num_elem() * sizeof(float), hipMemcpyDeviceToHost);

        // Print values of logit_a
        // printf("logit_a: ");
        // for (int i = 0; i < 15; i++) 
        //   printf("%f ", logit_a->buf[i]);
        // printf("\n");

        /* Greedy sampling (only last timestep is considered) */
        int next_token_id = top1_sampling(logit_a);
        printf("next_token_id: %d\n", next_token_id);

        /* Update input prompt and prompt size */
        input_prompt.push_back(next_token_id);
        prompt_size += 1;

        /* Store generated token to output */
        output[p * n_token + t] = next_token_id;

        /* Finalize activations for next token generation */
        free_activations();
      }
    }
  }
}