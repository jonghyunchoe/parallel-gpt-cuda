#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cmath>
#include <cstdio>
#include <vector>
#include <iostream>

#include "layer.h"
#include "model.h"

#define BATCH_SIZE 2048
#define NGPU 4  // Number of GPUs per node

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

#define MIN(a, b) (((a) < (b)) ? (a) : (b))

void print_device_pointer(float* d_ptr, size_t N, int gpu_id) {
  float* h_ptr = (float*)malloc(N * sizeof(float));
  hipSetDevice(gpu_id);
  hipMemcpy(h_ptr, d_ptr, N * sizeof(float), hipMemcpyDeviceToHost);
  for (size_t i = 0; i < N; i += 8) {
    for (size_t j = 0; j < 8; j++) {
      if ((i+j) < N)
        printf("%lf ", h_ptr[i+j]);
    }
    printf("\n");
  }
  printf("\n");
  free(h_ptr);
}

void print_device_pointer(int* d_ptr, size_t N, int gpu_id, char* flag) {
  int* h_ptr = (int*)malloc(N * sizeof(int));
  hipSetDevice(gpu_id);
  hipMemcpy(h_ptr, d_ptr, N * sizeof(int), hipMemcpyDeviceToHost);
  printf("Printing %s: \n", flag);
  for (size_t i = 0; i < N; i += 8) {
    for (size_t j = 0; j < 8; j++) {
      if ((i+j) < N)
        printf("%d ", h_ptr[i+j]);
    }
    printf("\n");
  }
  printf("\n");
  free(h_ptr);
}

// Parameters declaration
Parameter *attn_b[NUM_LAYER], *attn_w[NUM_LAYER];
Parameter *proj_b[NUM_LAYER], *proj_w[NUM_LAYER];
Parameter *ln_1_b[NUM_LAYER], *ln_1_g[NUM_LAYER];
Parameter *ln_2_b[NUM_LAYER], *ln_2_g[NUM_LAYER];
Parameter *mlp1_b[NUM_LAYER], *mlp1_w[NUM_LAYER];
Parameter *mlp2_b[NUM_LAYER], *mlp2_w[NUM_LAYER];
Parameter *ln_f_b, *ln_f_g;
Parameter *wpe, *wte;

float *d_attn_b[NUM_LAYER][NGPU], *d_attn_w[NUM_LAYER][NGPU];
float *d_proj_b[NUM_LAYER][NGPU], *d_proj_w[NUM_LAYER][NGPU];
float *d_ln_1_b[NUM_LAYER][NGPU], *d_ln_1_g[NUM_LAYER][NGPU];
float *d_ln_2_b[NUM_LAYER][NGPU], *d_ln_2_g[NUM_LAYER][NGPU];
float *d_mlp1_b[NUM_LAYER][NGPU], *d_mlp1_w[NUM_LAYER][NGPU];
float *d_mlp2_b[NUM_LAYER][NGPU], *d_mlp2_w[NUM_LAYER][NGPU];
float *d_ln_f_b[NGPU], *d_ln_f_g[NGPU];
float *d_wpe[NGPU], *d_wte[NGPU];

void alloc_and_set_device_parameters() {
  int order[] = {
      0, 1, 10, 11, 2, 3, 4, 5, 6, 7, 8, 9,
  };

  for (int gpu_id = 0; gpu_id < NGPU; gpu_id++) {
    hipSetDevice(gpu_id);

    for (int i = 0; i < NUM_LAYER; i++) {
      hipMalloc(&d_attn_b[order[i]][gpu_id], attn_b[order[i]]->num_elem() * sizeof(float));
      hipMalloc(&d_attn_w[order[i]][gpu_id], attn_w[order[i]]->num_elem() * sizeof(float));
      hipMalloc(&d_proj_b[order[i]][gpu_id], proj_b[order[i]]->num_elem() * sizeof(float));
      hipMalloc(&d_proj_w[order[i]][gpu_id], proj_w[order[i]]->num_elem() * sizeof(float));
      hipMalloc(&d_ln_1_b[order[i]][gpu_id], ln_1_b[order[i]]->num_elem() * sizeof(float));
      hipMalloc(&d_ln_1_g[order[i]][gpu_id], ln_1_g[order[i]]->num_elem() * sizeof(float));
      hipMalloc(&d_ln_2_b[order[i]][gpu_id], ln_2_b[order[i]]->num_elem() * sizeof(float));
      hipMalloc(&d_ln_2_g[order[i]][gpu_id], ln_2_g[order[i]]->num_elem() * sizeof(float));
      hipMalloc(&d_mlp1_b[order[i]][gpu_id], mlp1_b[order[i]]->num_elem() * sizeof(float));
      hipMalloc(&d_mlp1_w[order[i]][gpu_id], mlp1_w[order[i]]->num_elem() * sizeof(float));
      hipMalloc(&d_mlp2_b[order[i]][gpu_id], mlp2_b[order[i]]->num_elem() * sizeof(float));
      hipMalloc(&d_mlp2_w[order[i]][gpu_id], mlp2_w[order[i]]->num_elem() * sizeof(float));
    }
    hipMalloc(&d_ln_f_b[gpu_id], ln_f_b->num_elem() * sizeof(float));
    hipMalloc(&d_ln_f_g[gpu_id], ln_f_g->num_elem() * sizeof(float));
    hipMalloc(&d_wpe[gpu_id], wpe->num_elem() * sizeof(float));
    hipMalloc(&d_wte[gpu_id], wte->num_elem() * sizeof(float));
    
    // Copy data to device
    for (int i = 0; i < NUM_LAYER; i++) {
      hipMemcpy(d_attn_b[order[i]][gpu_id], attn_b[order[i]]->buf, attn_b[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_attn_w[order[i]][gpu_id], attn_w[order[i]]->buf, attn_w[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_proj_b[order[i]][gpu_id], proj_b[order[i]]->buf, proj_b[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_proj_w[order[i]][gpu_id], proj_w[order[i]]->buf, proj_w[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_ln_1_b[order[i]][gpu_id], ln_1_b[order[i]]->buf, ln_1_b[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_ln_1_g[order[i]][gpu_id], ln_1_g[order[i]]->buf, ln_1_g[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_ln_2_b[order[i]][gpu_id], ln_2_b[order[i]]->buf, ln_2_b[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_ln_2_g[order[i]][gpu_id], ln_2_g[order[i]]->buf, ln_2_g[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_mlp1_b[order[i]][gpu_id], mlp1_b[order[i]]->buf, mlp1_b[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_mlp1_w[order[i]][gpu_id], mlp1_w[order[i]]->buf, mlp1_w[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_mlp2_b[order[i]][gpu_id], mlp2_b[order[i]]->buf, mlp2_b[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_mlp2_w[order[i]][gpu_id], mlp2_w[order[i]]->buf, mlp2_w[order[i]]->num_elem() * sizeof(float), hipMemcpyHostToDevice);
    }
    hipMemcpy(d_ln_f_b[gpu_id], ln_f_b->buf, ln_f_b->num_elem() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ln_f_g[gpu_id], ln_f_g->buf, ln_f_g->num_elem() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_wpe[gpu_id], wpe->buf, wpe->num_elem() * sizeof(float), hipMemcpyHostToDevice); 
    hipMemcpy(d_wte[gpu_id], wte->buf, wte->num_elem() * sizeof(float), hipMemcpyHostToDevice);
  }
}

void alloc_and_set_parameters(float *param) {
  size_t pos = 0;
  int order[] = {
      0, 1, 10, 11, 2, 3, 4, 5, 6, 7, 8, 9,
  };
  for (int i = 0; i < NUM_LAYER; i++) {
    attn_b[order[i]] = new Parameter({3 * HIDDEN_DIM}, param + pos);
    pos += OFFSET1;
    attn_w[order[i]] = new Parameter({HIDDEN_DIM, 3 * HIDDEN_DIM}, param + pos);
    pos += OFFSET2;
    proj_b[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    proj_w[order[i]] = new Parameter({HIDDEN_DIM, HIDDEN_DIM}, param + pos);
    pos += OFFSET4;
    ln_1_b[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    ln_1_g[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    ln_2_b[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    ln_2_g[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    mlp1_b[order[i]] = new Parameter({4 * HIDDEN_DIM}, param + pos);
    pos += OFFSET5;
    mlp1_w[order[i]] = new Parameter({HIDDEN_DIM, 4 * HIDDEN_DIM}, param + pos);
    pos += OFFSET6;
    mlp2_b[order[i]] = new Parameter({HIDDEN_DIM}, param + pos);
    pos += OFFSET3;
    mlp2_w[order[i]] = new Parameter({4 * HIDDEN_DIM, HIDDEN_DIM}, param + pos);
    pos += OFFSET6;
  }
  ln_f_b = new Parameter({HIDDEN_DIM}, param + pos);
  pos += OFFSET3;
  ln_f_g = new Parameter({HIDDEN_DIM}, param + pos);
  pos += OFFSET3;
  wpe = new Parameter({MAX_SEQ_LEN, HIDDEN_DIM}, param + pos);
  pos += OFFSET7;
  wte = new Parameter({NUM_VOCAB, HIDDEN_DIM}, param + pos);
  pos += OFFSET8;

  alloc_and_set_device_parameters();
}

void free_parameters() {
  for (int gpu_id = 0; gpu_id < NGPU; gpu_id++) {
    hipSetDevice(gpu_id);

    for (int i = 0; i < NUM_LAYER; i++) {
      hipFree(d_attn_b[i][gpu_id]);
      hipFree(d_attn_w[i][gpu_id]);
      hipFree(d_proj_b[i][gpu_id]);
      hipFree(d_proj_w[i][gpu_id]);
      hipFree(d_ln_1_b[i][gpu_id]);
      hipFree(d_ln_1_g[i][gpu_id]);
      hipFree(d_ln_2_b[i][gpu_id]);
      hipFree(d_ln_2_g[i][gpu_id]);
      hipFree(d_mlp1_b[i][gpu_id]);
      hipFree(d_mlp1_w[i][gpu_id]);
      hipFree(d_mlp2_b[i][gpu_id]);
      hipFree(d_mlp2_w[i][gpu_id]);
    }
    hipFree(d_ln_f_b[gpu_id]);
    hipFree(d_ln_f_g[gpu_id]);
    hipFree(d_wpe[gpu_id]);
    hipFree(d_wte[gpu_id]);
  }
}

float *d_embd_a[NGPU], *d_ffn_proj_a[NGPU];
float *d_mha_qkv_proj_a[NGPU], *d_mha_out_a[NGPU], *d_mha_split_qkv_a[NGPU],
    *d_mha_split_head_a[NGPU], *d_mha_mask_a[NGPU], *d_mha_merge_head_a[NGPU], *d_mha_q_a[NGPU],
    *d_mha_k_a[NGPU], *d_mha_v_a[NGPU], *d_mha_attn_out_a[NGPU], *d_mha_concat_head_a[NGPU];
float *d_attn_score_a[NGPU], *d_k_transposed_a[NGPU];
float *d_wte_transposed_a[NGPU], *d_residual_a[NGPU], *d_logit_a[NGPU];
float *d_transformer_block_a[NGPU];

void alloc_activations(size_t prompt_size, int gpu_id) {
  hipSetDevice(gpu_id);

  hipMalloc(&d_embd_a[gpu_id], prompt_size * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_ffn_proj_a[gpu_id], prompt_size * 4 * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_mha_qkv_proj_a[gpu_id], prompt_size * 3 * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_mha_out_a[gpu_id], prompt_size * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_mha_split_qkv_a[gpu_id], 3 * prompt_size * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_mha_split_head_a[gpu_id], 3 * NUM_HEAD * prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_mask_a[gpu_id], prompt_size * prompt_size * sizeof(float));
  hipMalloc(&d_mha_merge_head_a[gpu_id], NUM_HEAD * prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_q_a[gpu_id], prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_k_a[gpu_id], prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_v_a[gpu_id], prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_attn_out_a[gpu_id], prompt_size * HIDDEN_DIM / NUM_HEAD * sizeof(float));
  hipMalloc(&d_mha_concat_head_a[gpu_id], prompt_size * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_attn_score_a[gpu_id], prompt_size * prompt_size * sizeof(float));
  hipMalloc(&d_k_transposed_a[gpu_id], HIDDEN_DIM / NUM_HEAD * prompt_size * sizeof(float));
  hipMalloc(&d_wte_transposed_a[gpu_id], HIDDEN_DIM * NUM_VOCAB * sizeof(float));
  hipMalloc(&d_residual_a[gpu_id], prompt_size * HIDDEN_DIM * sizeof(float));
  hipMalloc(&d_logit_a[gpu_id], prompt_size * NUM_VOCAB * sizeof(float));
  hipMalloc(&d_transformer_block_a[gpu_id], prompt_size * HIDDEN_DIM * sizeof(float));
}

void free_activations(int gpu_id) {
  hipSetDevice(gpu_id);

  hipFree(d_embd_a[gpu_id]);
  hipFree(d_ffn_proj_a[gpu_id]);
  hipFree(d_mha_qkv_proj_a[gpu_id]);
  hipFree(d_mha_out_a[gpu_id]);
  hipFree(d_mha_split_qkv_a[gpu_id]);
  hipFree(d_mha_split_head_a[gpu_id]);
  hipFree(d_mha_mask_a[gpu_id]);
  hipFree(d_mha_merge_head_a[gpu_id]);
  hipFree(d_mha_q_a[gpu_id]);
  hipFree(d_mha_k_a[gpu_id]);
  hipFree(d_mha_v_a[gpu_id]);
  hipFree(d_mha_attn_out_a[gpu_id]);
  hipFree(d_mha_concat_head_a[gpu_id]);
  hipFree(d_attn_score_a[gpu_id]);
  hipFree(d_k_transposed_a[gpu_id]);
  hipFree(d_wte_transposed_a[gpu_id]);
  hipFree(d_residual_a[gpu_id]);
  hipFree(d_logit_a[gpu_id]);
  hipFree(d_transformer_block_a[gpu_id]);
}

// Adapted functions for multi-GPU
void ffn(float *d_in, float *d_mlp1_w, float *d_mlp1_b,
         float *d_mlp2_w, float *d_mlp2_b, float *d_out, size_t seq_len, size_t batch_size, int gpu_id) {
    hipSetDevice(gpu_id);

    batch_linear(d_in, d_mlp1_w, d_mlp1_b, d_ffn_proj_a[gpu_id], batch_size, seq_len, HIDDEN_DIM, 4 * HIDDEN_DIM);
    batch_gelu(d_ffn_proj_a[gpu_id], batch_size, seq_len * 4 * HIDDEN_DIM);
    batch_linear(d_ffn_proj_a[gpu_id], d_mlp2_w, d_mlp2_b, d_out, batch_size, seq_len, 4 * HIDDEN_DIM, HIDDEN_DIM);
}

void attention(float *d_q, float *d_k, float *d_v, float *d_mask, float *d_out, size_t seq_len, size_t head_dim, size_t batch_size, int gpu_id) {
    hipSetDevice(gpu_id);

    batch_transpose(d_k, d_k_transposed_a[gpu_id], batch_size, seq_len, head_dim);
    batch_matmul(d_q, d_k_transposed_a[gpu_id], d_attn_score_a[gpu_id], batch_size, seq_len, head_dim, seq_len);
    batch_scaling(d_attn_score_a[gpu_id], 1.0 / sqrt(head_dim), batch_size, seq_len * seq_len);
    batch_add(d_attn_score_a[gpu_id], d_mask, batch_size, seq_len * seq_len);
    batch_softmax(d_attn_score_a[gpu_id], batch_size, seq_len, seq_len);
    batch_matmul(d_attn_score_a[gpu_id], d_v, d_out, batch_size, seq_len, seq_len, head_dim);
}

void mha(float *d_in, float *d_attn_b, float *d_attn_w,
         float *d_proj_b, float *d_proj_w, float *d_out, size_t seq_len, size_t batch_size, int gpu_id) {
    hipSetDevice(gpu_id);

    batch_linear(d_in, d_attn_w, d_attn_b, d_mha_qkv_proj_a[gpu_id], batch_size, seq_len, HIDDEN_DIM, 3 * HIDDEN_DIM);
    batch_split_qkv(d_mha_qkv_proj_a[gpu_id], d_mha_split_qkv_a[gpu_id], batch_size, seq_len, 3 * HIDDEN_DIM);
    batch_split_head(d_mha_split_qkv_a[gpu_id], d_mha_split_head_a[gpu_id], batch_size, NUM_HEAD, seq_len, HIDDEN_DIM);
    batch_generate_mask(d_mha_mask_a[gpu_id], batch_size, seq_len);

    for (size_t idx = 0; idx < NUM_HEAD; idx++) {
        batch_extract_qkv(d_mha_split_head_a[gpu_id], d_mha_q_a[gpu_id], d_mha_k_a[gpu_id], d_mha_v_a[gpu_id], batch_size, idx, NUM_HEAD, seq_len, HIDDEN_DIM / NUM_HEAD);
        attention(d_mha_q_a[gpu_id], d_mha_k_a[gpu_id], d_mha_v_a[gpu_id], d_mha_mask_a[gpu_id], d_mha_attn_out_a[gpu_id], seq_len, HIDDEN_DIM / NUM_HEAD, batch_size, gpu_id);
        batch_merge_head(d_mha_attn_out_a[gpu_id], d_mha_merge_head_a[gpu_id], batch_size, NUM_HEAD, idx, seq_len, HIDDEN_DIM / NUM_HEAD);
    }

    batch_concat_head(d_mha_merge_head_a[gpu_id], d_mha_concat_head_a[gpu_id], batch_size, NUM_HEAD, seq_len, HIDDEN_DIM / NUM_HEAD);
    batch_linear(d_mha_concat_head_a[gpu_id], d_proj_w, d_proj_b, d_out, batch_size, seq_len, HIDDEN_DIM, HIDDEN_DIM);
}

void transformer_block(float *d_in, float *d_attn_b, float *d_attn_w,
                       float *d_proj_b, float *d_proj_w, float *d_ln_1_b,
                       float *d_ln_1_g, float *d_ln_2_b, float *d_ln_2_g,
                       float *d_mlp1_b, float *d_mlp1_w, float *d_mlp2_b,
                       float *d_mlp2_w, float *d_out, size_t seq_len, size_t batch_size, int gpu_id) { 
    hipSetDevice(gpu_id);

    batch_copy(d_in, d_residual_a[gpu_id], batch_size, seq_len * HIDDEN_DIM);
    batch_layer_norm(d_in, d_ln_1_g, d_ln_1_b, batch_size, seq_len, HIDDEN_DIM, 1e-5);
    mha(d_in, d_attn_b, d_attn_w, d_proj_b, d_proj_w, d_mha_out_a[gpu_id], seq_len, batch_size, gpu_id);
    batch_add(d_mha_out_a[gpu_id], d_residual_a[gpu_id], batch_size, seq_len * HIDDEN_DIM);
    batch_copy(d_mha_out_a[gpu_id], d_residual_a[gpu_id], batch_size, seq_len * HIDDEN_DIM);
    batch_layer_norm(d_mha_out_a[gpu_id], d_ln_2_g, d_ln_2_b, batch_size, seq_len, HIDDEN_DIM, 1e-5);
    ffn(d_mha_out_a[gpu_id], d_mlp1_w, d_mlp1_b, d_mlp2_w, d_mlp2_b, d_out, seq_len, batch_size, gpu_id);
    batch_add(d_out, d_residual_a[gpu_id], batch_size, seq_len * HIDDEN_DIM);
}

__global__ void insert_tokens_kernel(int *d_out, int *d_input_prompt, int *d_buffer, int prompt_size, int n_token, int batch_size, int position, int total_size) {
    // TODO update 
    // int total_size = batch_size * (prompt_size + batch_size);

    // Overwriting to d_out here 
    // Maybe something with temp? 
    // TODO allocate separate temp device pointer and do hipMalloc and pass pointer to here 
    // temp device pointer should be for ngpus 
    // *d_buffer[ngpu] would work

    // int *temp = new int[total_size];

    for (int i = 0; i < batch_size; i++) {
        for (int j = 0; j < prompt_size; j++) {
            // 첫번째 prompt는 그대로 
            // 두번째 prompt는 원래에서 + 1 
            // 세번째 prompt는 원래에서 + 2 
            // ... 
            d_buffer[i * prompt_size + i + j] = d_input_prompt[i * prompt_size + j];
        }
    }

    for (int i = 0; i < batch_size; i++) {
        // 첫번째는 첫번째 prompt의 끝 (prompt_size)
        // 두번째는 두번째 prompt의 끝 (prompt_size * 2 + 1)
        // 세번째는 세번째 prompt의 끝 (prompt_size * 3 + 2)
        int insert_position = (i + 1) * prompt_size + i;
        // switch i + position to i * prompt_size + position where position is nth token 
        // temp[insert_position] = d_out[i + position];
        d_buffer[insert_position] = d_out[i * n_token + position];
        // printf("Inserting token %d at position %d from i + position %d\n", d_out[i * n_token + position], insert_position, i * n_token + position);
    }

    for (int i = 0; i < total_size; i++) {
        d_input_prompt[i] = d_buffer[i];
    }
}


void generate_tokens(int *input, int *output, size_t n_prompt, size_t n_token) {
    int mpi_rank, mpi_size;
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);

    printf("\n");
    if (mpi_rank != 0) {
        input = (int *)malloc(n_prompt * tokens_per_prompt * sizeof(int));
        output = (int *)malloc(n_prompt * n_token * sizeof(int));
    }

    MPI_Bcast(input, n_prompt * tokens_per_prompt, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(output, n_prompt * n_token, MPI_INT, 0, MPI_COMM_WORLD);

    size_t prompts_per_node = (n_prompt + mpi_size - 1) / mpi_size;
    size_t start_prompt = mpi_rank * prompts_per_node;
    size_t end_prompt = MIN(start_prompt + prompts_per_node, n_prompt);

    for (size_t p = start_prompt; p < end_prompt; p += BATCH_SIZE) {
        int batch_size = MIN(BATCH_SIZE, end_prompt - p);
        int prompt_size = tokens_per_prompt;

        std::vector<int> input_prompt(batch_size * prompt_size);
        memcpy(input_prompt.data(), input + p * prompt_size, batch_size * prompt_size * sizeof(int));

        int *d_input_prompt[NGPU];
        int *d_out[NGPU];
        int *d_buffer[NGPU]; 

        for (int gpu_id = 0; gpu_id < NGPU; gpu_id++) {
            hipSetDevice(gpu_id);
            size_t start_idx = gpu_id * (batch_size / NGPU);
            size_t end_idx = (gpu_id == NGPU - 1) ? batch_size : (gpu_id + 1) * (batch_size / NGPU);
            size_t gpu_batch_size = end_idx - start_idx;

            // TODO check if d_input_prompt has to be allocated gpu_batch_size instead of batch_size
            CHECK_CUDA(hipMalloc(&d_input_prompt[gpu_id], batch_size * (prompt_size + n_token - 1) * sizeof(int)));
            CHECK_CUDA(hipMalloc(&d_buffer[gpu_id], batch_size * (prompt_size + n_token - 1) * sizeof(int)));
            // TODO allocate batch_size * n_token as d_out will contain n_token tokens for each batch
            CHECK_CUDA(hipMalloc(&d_out[gpu_id], batch_size * n_token * sizeof(int)));
            alloc_activations(batch_size * (prompt_size + n_token - 1), gpu_id);
            // Copy input_prompt to d_input_prompt before token generation loop
            CHECK_CUDA(hipMemcpy(d_input_prompt[gpu_id], input_prompt.data() + start_idx * prompt_size, gpu_batch_size * prompt_size * sizeof(int), hipMemcpyHostToDevice));
        }

        for (size_t t = 0; t < n_token; t++) {
            for (int gpu_id = 0; gpu_id < NGPU; gpu_id++) {
                hipSetDevice(gpu_id);
                size_t start_idx = gpu_id * (batch_size / NGPU);
                size_t end_idx = (gpu_id == NGPU - 1) ? batch_size : (gpu_id + 1) * (batch_size / NGPU);
                size_t gpu_batch_size = end_idx - start_idx;

                // CHECK_CUDA(hipMemcpy(d_input_prompt[gpu_id], input_prompt.data() + start_idx * prompt_size, gpu_batch_size * prompt_size * sizeof(int), hipMemcpyHostToDevice));
                
                // Print input prompt
                // print_device_pointer(d_input_prompt[gpu_id], gpu_batch_size * prompt_size, gpu_id, "d_input_prompt at start of token generation loop");

                // print_device_pointer(d_out[gpu_id], gpu_batch_size * n_token, gpu_id, "d_out at start of token generation loop");

                batch_token_pos_embedding(d_input_prompt[gpu_id], d_wte[gpu_id], d_wpe[gpu_id], d_embd_a[gpu_id], gpu_batch_size, prompt_size, HIDDEN_DIM);

                for (size_t l = 0; l < NUM_LAYER; l++) {
                    transformer_block(d_embd_a[gpu_id], d_attn_b[l][gpu_id], d_attn_w[l][gpu_id], d_proj_b[l][gpu_id], d_proj_w[l][gpu_id],
                                      d_ln_1_b[l][gpu_id], d_ln_1_g[l][gpu_id], d_ln_2_b[l][gpu_id], d_ln_2_g[l][gpu_id],
                                      d_mlp1_b[l][gpu_id], d_mlp1_w[l][gpu_id], d_mlp2_b[l][gpu_id], d_mlp2_w[l][gpu_id],
                                      d_transformer_block_a[gpu_id], prompt_size, gpu_batch_size, gpu_id);
                    batch_copy(d_transformer_block_a[gpu_id], d_embd_a[gpu_id], gpu_batch_size, prompt_size * HIDDEN_DIM);
                }

                batch_layer_norm(d_embd_a[gpu_id], d_ln_f_g[gpu_id], d_ln_f_b[gpu_id], gpu_batch_size, prompt_size, HIDDEN_DIM, 1e-5);
                transpose(d_wte[gpu_id], d_wte_transposed_a[gpu_id], wte->shape[0], wte->shape[1]);
                batch_matmul_final(d_embd_a[gpu_id], d_wte_transposed_a[gpu_id], d_logit_a[gpu_id], gpu_batch_size, prompt_size, HIDDEN_DIM, wte->shape[0]);
                // TODO add d_out to next batch_size index instead of starting from 0 
                // TODO change name from position to more appropriate name 
                // print_device_pointer(d_out[gpu_id], gpu_batch_size * n_token, gpu_id, "d_out before top1_sampling");
                // batch_top1_sampling(d_logit_a[gpu_id], d_out[gpu_id], gpu_batch_size, gpu_batch_size * t, prompt_size, NUM_VOCAB);
                batch_top1_sampling(d_logit_a[gpu_id], d_out[gpu_id], gpu_batch_size, n_token, t, prompt_size, NUM_VOCAB);

                // print_device_pointer(d_out[gpu_id], gpu_batch_size * n_token, gpu_id, "d_out after top1_sampling");
                // Insert tokens directly from d_out to d_input_prompt
                // print_device_pointer(d_input_prompt[gpu_id], gpu_batch_size * (prompt_size + t + 1), gpu_id, "d_input_prompt before inserting tokens");
                // TODO move previous input_prompt a step away 
                // insert_tokens_kernel<<<1, 1>>>(d_out[gpu_id], d_input_prompt[gpu_id], prompt_size, gpu_batch_size, gpu_batch_size * t, batch_size * (prompt_size + n_token - 1));
                insert_tokens_kernel<<<1, 1>>>(d_out[gpu_id], d_input_prompt[gpu_id], d_buffer[gpu_id], prompt_size, n_token, gpu_batch_size, t, batch_size * (prompt_size + n_token - 1));
                // print_device_pointer(d_out[gpu_id], gpu_batch_size * n_token, gpu_id, "d_out after inserting token");
                // print_device_pointer(d_input_prompt[gpu_id], gpu_batch_size * (prompt_size + t + 1), gpu_id, "d_input_prompt after inserting tokens");
            }

            prompt_size += 1;
            // if (t == 4)
            //   exit(1);
        }

        // for (int gpu_id = 0; gpu_id < NGPU; gpu_id++) {
        //     hipSetDevice(gpu_id);
        //     CHECK_CUDA(hipFree(d_input_prompt[gpu_id]));
        //     CHECK_CUDA(hipFree(d_out[gpu_id]));
        //     free_activations(gpu_id);
        // }

        // TODO write from d_output to output 
        for (int gpu_id = 0; gpu_id < NGPU; gpu_id++) {
            hipSetDevice(gpu_id); 

            // For logging 
            // TODO perhaps I should write to d_out first prompt's output and then second prompt's output instead of interleaving 
            size_t start_idx = gpu_id * (batch_size / NGPU);
            size_t end_idx = (gpu_id == NGPU - 1) ? batch_size : (gpu_id + 1) * (batch_size / NGPU);
            size_t gpu_batch_size = end_idx - start_idx;
            // print_device_pointer(d_out[gpu_id], gpu_batch_size * n_token, gpu_id, "d_out before copying into output");
            CHECK_CUDA(hipMemcpy(output + p * n_token + start_idx * n_token, d_out[gpu_id], gpu_batch_size * n_token * sizeof(int), hipMemcpyDeviceToHost));
            
            // Print output 
            // for (size_t i = 0; i < gpu_batch_size; i++) {
            //     printf("Output for prompt %zu: ", p + i);
            //     for (size_t j = 0; j < n_token; j++) {
            //         printf("%d ", output[(p + i) * n_token + j]);
            //     }
            //     printf("\n");
            // }
            // printf("\n");

            // Print output 
            // print_device_pointer(d_out[gpu_id], batch_size * n_token, gpu_id, "d_out after token generation loop");
        }
    }

    if (mpi_rank == 0) {
        std::vector<int> final_output(n_prompt * n_token);
        MPI_Gather(output, prompts_per_node * n_token, MPI_INT, final_output.data(), prompts_per_node * n_token, MPI_INT, 0, MPI_COMM_WORLD);
        memcpy(output, final_output.data(), n_prompt * n_token * sizeof(int));
    } else {
        MPI_Gather(output + start_prompt * n_token, prompts_per_node * n_token, MPI_INT, NULL, 0, MPI_INT, 0, MPI_COMM_WORLD);
    }

    // Print output
    if (mpi_rank == 0) {
        for (size_t i = 0; i < 8; i++) {
            printf("Prompt %zu: ", i);
            for (size_t j = 0; j < tokens_per_prompt; j++) {
                printf("%d ", input[i * tokens_per_prompt + j]);
            }
            printf("\n");

            printf("Output %zu: ", i);
            for (size_t j = 0; j < n_token; j++) {
                printf("%d ", output[i * n_token + j]);
            }
            printf("\n");
        }
    }
}
