#include "hip/hip_runtime.h"
#include "layer.h"
#include <cstdio>

#define TILE_SIZE 32 

/* Token + Positional Embedding
 * @param [in1]  in: [s]
 * @param [in2] wte: [NUM_VOCAB, H]
 * @param [in3] wpe: [MAX_SEQ_LEN, H]
 * @param [out] out: [s, H]
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 */
// void token_pos_embedding(vector<int> in, Tensor *wte, Tensor *wpe,
//                          Tensor *out) {
//   size_t s = in.size();
//   size_t H = wte->shape[1];

//   for (size_t i = 0; i < s; i++) {
//     for (size_t j = 0; j < H; j++) {
//       out->buf[i * H + j] = wte->buf[in[i] * H + j] + wpe->buf[i * H + j];
//     }
//   }
// }

__global__ void token_pos_embedding_kernel(int *in, float *wte, float *wpe, float *out, size_t s, size_t H) {
  // printf("Start\n");
  // printf("s * H: %d\n", s * H);
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x; 
  // printf("idx: %d\n", idx);
  // printf("H: %d\n", H);

  if (idx < s * H) {
    // printf("Within if\n");
    // printf("s * H: %d\n", s * H);
    // printf("H: %zu\n", H);
    // printf("On DEV: Size: %lu, Minus One: %d\n",(unsigned long)(sp->size), (int)-1);
    // printf("%lu\n",(unsigned long)(H));

    // printf("idx: %lu, H: %lu\n", (unsigned long)idx, (unsigned long)H);

    // printf("idx: %d, H: %d\n", idx, H);
    // printf("H: %d\n", H);
    size_t i = idx / H; 
    size_t j = idx % H; 
    // printf("i: %lu, j: %lu, idx: %lu, H: %lu\n", (unsigned long)i, (unsigned long)j, (unsigned long)idx, (unsigned long)H);

    // printf("wte[in[i] * H + j]: %f\n", wte[in[i] * H + j]);
    // printf("wpe[i * H + j]: %f\n", wpe[i * H + j]);
    // printf("H: %d\n", H);

    // printf("idx: %lu, H: %lu\n", (unsigned long)idx, (unsigned long)H);

    // printf("in[i]: %d\n", in[i]);
    // printf("in[i] * H + j: %lu\n", (unsigned long)(in[i] * H + j));
    // printf("wte[in[i] * H + j]: %f\n", wte[in[i] * H + j]);
    // printf("wpe[i * H + j]: %f\n", wpe[i * H + j]);
    // printf("out[idx]: %f\n", out[idx])
    out[idx] = wte[in[i] * H + j] + wpe[i * H + j]; 
    // printf("out[%lu]: %f\n", (unsigned long)idx, out[idx]);
  }
  // printf("out[%d]: %f\n", idx, out[idx]);
  // printf("End\n");
}

void token_pos_embedding(vector<int> in, Tensor *wte, Tensor *wpe, Tensor *out) {
  size_t s = in.size(); 
  size_t H = wte->shape[1]; 
  printf("wte->num_elem(): %lu\n", (unsigned long)(wte->num_elem()));

  int *d_in; 
  float *d_wte; 
  float *d_wpe; 
  float *d_out; 

  hipMalloc(&d_in, s * sizeof(int));
  hipMalloc(&d_wte, wte->num_elem() * sizeof(float));
  hipMalloc(&d_wpe, wpe->num_elem() * sizeof(float)); 
  hipMalloc(&d_out, s * H * sizeof(float));

  hipMemcpy(d_in, in.data(), s * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_wte, wte->buf, wte->num_elem() * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_wpe, wpe->buf, wpe->num_elem() * sizeof(float), hipMemcpyHostToDevice); 

  dim3 blockDim(256); 
  dim3 gridDim((s * H + blockDim.x - 1) / blockDim.x); 
  token_pos_embedding_kernel<<<gridDim, blockDim>>>(d_in, d_wte, d_wpe, d_out, s, H);

  hipMemcpy(out->buf, d_out, s * H * sizeof(float), hipMemcpyDeviceToHost); 

  hipFree(d_in);
  hipFree(d_wte);
  hipFree(d_wpe);
  hipFree(d_out);
}

void token_pos_embedding(int *d_in, float *d_wte, float *d_wpe, float *d_out, size_t s, size_t H) {
  printf("s: %zu, H: %zu\n", s, H);
  
  dim3 blockDim(256);
  dim3 gridDim((s * H + blockDim.x - 1) / blockDim.x); 
  token_pos_embedding_kernel<<<gridDim, blockDim>>>(d_in, d_wte, d_wpe, d_out, s, H);
  // printf("End\n");

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch token_pos_embedding_kernel (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}


/* GELU
 * @param [in & out] inout: [N]
 * 'N' is the number of elements in the tensor.
 */
// void gelu(Tensor *inout) {
//   size_t N = inout->num_elem();

//   for (size_t i = 0; i < N; i++) {
//     float x = inout->buf[i];
//     inout->buf[i] =
//         0.5 * x *
//         (1.f + tanh(sqrt(2.f / MATH_PI) * (x + 0.044715f * x * x * x)));
//   }
// }

__global__ void gelu_kernel(float *inout, size_t N) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x; 
  
  if (idx < N) {
    float x = inout[idx]; 
    inout[idx] = 
        0.5 * x *
        (1.f + tanh(sqrt(2.f / MATH_PI) * (x + 0.044715f * x * x * x)));
  }
}

void gelu(Tensor *inout) {
  size_t N = inout->num_elem(); 

  float *d_inout;

  hipMalloc(&d_inout, N * sizeof(float));

  hipMemcpy(d_inout, inout->buf, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(256); 
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x); 
  gelu_kernel<<<gridDim, blockDim>>>(d_inout, N);

  hipMemcpy(inout->buf, d_inout, N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_inout);
}

void gelu(float *d_inout, size_t N) {
    dim3 blockDim(256); 
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x); 
    gelu_kernel<<<gridDim, blockDim>>>(d_inout, N);
}


/* Softmax (w/ Max Trick)
 * @param [in & out] inout: [s, H]
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 */
// void softmax(Tensor *inout) {
//   size_t s = inout->shape[0];
//   size_t H = inout->shape[1];

//   for (size_t i = 0; i < s; i++) {
//     float max_val = inout->buf[i * H];
//     for (size_t j = 0; j < H; j++) {
//       if (inout->buf[i * H + j] > max_val) { max_val = inout->buf[i * H + j]; }
//     }

//     float sum = 0;
//     for (size_t j = 0; j < H; j++) {
//       inout->buf[i * H + j] = exp(inout->buf[i * H + j] - max_val);
//       sum += inout->buf[i * H + j];
//     }

//     for (size_t j = 0; j < H; j++) { inout->buf[i * H + j] /= sum; }
//   }
// }

__global__ void softmax_kernel(float *inout, size_t s, size_t V) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (idx < s) {
    float max_val = -INFINITY;
    for (size_t j = 0; j < V; j++) {
      max_val = fmaxf(max_val, inout[idx * V + j]);
    }

    float sum_exp = 0.0f;
    for (size_t j = 0; j < V; j++) {
      sum_exp += expf(inout[idx * V + j] - max_val);
    }

    for (size_t j = 0; j < V; j++) {
      inout[idx * V + j] = expf(inout[idx * V + j] - max_val) / sum_exp;
    }
  }
}

void softmax(Tensor *inout) {
  size_t s = inout->shape[0];
  size_t V = inout->shape[1];

  float *d_inout;

  hipMalloc(&d_inout, s * V * sizeof(float));

  hipMemcpy(d_inout, inout->buf, s * V * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(256);
  dim3 gridDim((s + blockDim.x - 1) / blockDim.x);
  softmax_kernel<<<gridDim, blockDim>>>(d_inout, s, V);

  hipMemcpy(inout->buf, d_inout, s * V * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_inout);
}

void softmax(float *d_inout, size_t s, size_t V) {
    dim3 blockDim(256);
    dim3 gridDim((s + blockDim.x - 1) / blockDim.x);
    softmax_kernel<<<gridDim, blockDim>>>(d_inout, s, V);
}

/* Layer Normalization
 * @param [in1 & out] inout: [s, H]
 * @param [in2]       gamma: [H]
 * @param [in3]        beta: [H]
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 */
// void layer_norm(Tensor *inout, Tensor *gamma, Tensor *beta) {
//   size_t s = inout->shape[0];
//   size_t H = inout->shape[1];

//   float eps = 1e-5;
//   for (size_t i = 0; i < s; i++) {
//     float mean = 0;
//     float var = 0;

//     for (size_t j = 0; j < H; j++) {
//       mean += inout->buf[i * H + j];
//       var += inout->buf[i * H + j] * inout->buf[i * H + j];
//     }

//     mean /= H;
//     var = var / H - mean * mean;

//     for (size_t j = 0; j < H; j++) {
//       inout->buf[i * H + j] = (inout->buf[i * H + j] - mean) *
//                                   (1.0 / sqrt(var + eps)) * gamma->buf[j] +
//                               beta->buf[j];
//     }
//   }
// }

__global__ void layer_norm_kernel(float *inout, float *gamma, float *beta, size_t s, size_t H, float eps) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < s) {
    float mean = 0;
    float var = 0;

    for (size_t j = 0; j < H; j++) {
      mean += inout[i * H + j];
      var += inout[i * H + j] * inout[i * H + j];
    }
    mean /= H;
    var = var / H - mean * mean;

    for (size_t j = 0; j < H; j++) {
      inout[i * H + j] = (inout[i * H + j] - mean) * (1.0 / sqrtf(var + eps)) * gamma[j] + beta[j];
    }
  }
}

void layer_norm(Tensor *inout, Tensor *gamma, Tensor *beta) {
  size_t s = inout->shape[0];
  size_t H = inout->shape[1];

  float eps = 1e-5;
  float *d_inout;
  float *d_gamma;
  float *d_beta;

  hipMalloc(&d_inout, s * H * sizeof(float));
  hipMalloc(&d_gamma, H * sizeof(float));
  hipMalloc(&d_beta, H * sizeof(float));

  hipMemcpy(d_inout, inout->buf, s * H * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_gamma, gamma->buf, H * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_beta, beta->buf, H * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(256);
  dim3 gridDim((s + blockDim.x - 1) / blockDim.x);
  layer_norm_kernel<<<gridDim, blockDim>>>(d_inout, d_gamma, d_beta, s, H, eps);

  hipMemcpy(inout->buf, d_inout, s * H * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_inout);
  hipFree(d_gamma);
  hipFree(d_beta);
}

void layer_norm(float *d_inout, float *d_gamma, float *d_beta, size_t s, size_t H, float eps) {
    dim3 blockDim(256);
    dim3 gridDim((s + blockDim.x - 1) / blockDim.x);
    layer_norm_kernel<<<gridDim, blockDim>>>(d_inout, d_gamma, d_beta, s, H, eps);
}

/* Linear
 * @param [in1]  in: [M, K]
 * @param [in2]   w: [K, N]
 * @param [in3]   b: [N]
 * @param [out] out: [M, N]
 */
// void linear(Tensor *in, Tensor *w, Tensor *b, Tensor *out) {
//   size_t M = in->shape[0];
//   size_t K = in->shape[1];
//   size_t N = w->shape[1];

// #pragma omp parallel for
//   for (size_t i = 0; i < M; i++) {
//     for (size_t j = 0; j < N; j++) {
//       out->buf[i * N + j] = 0;
//       for (size_t k = 0; k < K; k++) {
//         out->buf[i * N + j] += in->buf[i * K + k] * w->buf[k * N + j];
//       }
//       out->buf[i * N + j] += b->buf[j];
//     }
//   }
// }

__global__ void linear_kernel(float *in, float *w, float *b, float *out, size_t M, size_t K, size_t N) {
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < M && col < N) {
    float sum = 0;
    for (size_t k = 0; k < K; k++) {
      sum += in[row * K + k] * w[k * N + col];
    }
    out[row * N + col] = sum + b[col];
  }
}

void linear(Tensor *in, Tensor *w, Tensor *b, Tensor *out) {
  size_t M = in->shape[0];
  size_t K = in->shape[1];
  size_t N = w->shape[1];

  float *d_in;
  float *d_w;
  float *d_b;
  float *d_out;

  hipMalloc(&d_in, M * K * sizeof(float));
  hipMalloc(&d_w, K * N * sizeof(float));
  hipMalloc(&d_b, N * sizeof(float));
  hipMalloc(&d_out, M * N * sizeof(float));

  hipMemcpy(d_in, in->buf, M * K * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w, w->buf, K * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b->buf, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(16, 16);
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

  linear_kernel<<<gridDim, blockDim>>>(d_in, d_w, d_b, d_out, M, K, N);

  hipMemcpy(out->buf, d_out, M * N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_w);
  hipFree(d_b);
  hipFree(d_out);
}

void linear(float *d_in, float *d_w, float *d_b, float *d_out, size_t M, size_t K, size_t N) {
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);
    linear_kernel<<<gridDim, blockDim>>>(d_in, d_w, d_b, d_out, M, K, N);
}

/* Matmul
 * @param [in1]  in1: [M, K]
 * @param [in2]  in2: [K, N]
 * @param [out]  out: [M, N]
 */
// void matmul(Tensor *in1, Tensor *in2, Tensor *out) {
//   size_t M = in1->shape[0];
//   size_t K = in1->shape[1];
//   size_t N = in2->shape[1];

// #pragma omp parallel for
//   for (size_t i = 0; i < M; i++) {
//     for (size_t j = 0; j < N; j++) {
//       out->buf[i * N + j] = 0;
//       for (size_t k = 0; k < K; k++) {
//         out->buf[i * N + j] += in1->buf[i * K + k] * in2->buf[k * N + j];
//       }
//     }
//   }
// }

__global__ void matmul_kernel_naive(float *in1, float *in2, float *out, size_t M, size_t K, size_t N) {
  size_t row = blockIdx.y * blockDim.y + threadIdx.y; 
  size_t col = blockIdx.x * blockDim.x + threadIdx.x; 
  
  if (row < M && col < N) {
    float sum = 0;
    for (size_t k = 0; k < K; k++) {
      sum += in1[row * K + k] * in2[k * N + col];
    }
    out[row * N + col] = sum; 
  }
}

__global__ void matmul_kernel(float *A, float *B, float *C, size_t M, size_t K, size_t N) {
  __shared__ float A_shared[TILE_SIZE][TILE_SIZE];
  __shared__ float B_shared[TILE_SIZE][TILE_SIZE];   

  int global_row = blockIdx.y * blockDim.y + threadIdx.y;
  int global_col = blockIdx.x * blockDim.x + threadIdx.x; 
  int local_row = threadIdx.y;
  int local_col = threadIdx.x; 

  float sum = 0.0f;

  for (int k = 0; k < (K + TILE_SIZE - 1) / TILE_SIZE; k++) {
    int A_local_row = global_row * K + k * TILE_SIZE + local_col; 
    int B_local_col = (k * TILE_SIZE + local_row) * N + global_col; 
    
    if (global_row < M && k * TILE_SIZE + local_col < K) {
      A_shared[local_row][local_col] = A[A_local_row];
    } else {
      A_shared[local_row][local_col] = 0.0f; 
    }

    if (global_col < N && k * TILE_SIZE + local_row < K) {
      B_shared[local_row][local_col] = B[B_local_col];
    } else {
      B_shared[local_row][local_col] = 0.0f; 
    }

    __syncthreads();

    for (int n = 0; n < TILE_SIZE; n++) {
      sum += A_shared[local_row][n] * B_shared[n][local_col];
    }

    __syncthreads(); 
  }

  if (global_row < M && global_col < N) {
    C[global_row * N + global_col] = sum; 
  }
}

void matmul(Tensor *in1, Tensor *in2, Tensor *out) {
  size_t M = in1->shape[0]; 
  size_t K = in1->shape[1]; 
  size_t N = in2->shape[1]; 

  float *d_in1; 
  float *d_in2; 
  float *d_out; 

  hipMalloc(&d_in1, M * K * sizeof(float)); 
  hipMalloc(&d_in2, K * N * sizeof(float)); 
  hipMalloc(&d_out, M * N * sizeof(float)); 

  hipMemcpy(d_in1, in1->buf, M * K * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2->buf, K * N * sizeof(float), hipMemcpyHostToDevice); 

  dim3 blockDim(TILE_SIZE, TILE_SIZE);
  dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
  
  matmul_kernel<<<gridDim, blockDim>>>(d_in1, d_in2, d_out, M, K, N); 

  hipMemcpy(out->buf, d_out, M * N * sizeof(float), hipMemcpyDeviceToHost); 

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);
}

void matmul(float *d_in1, float *d_in2, float *d_out, size_t M, size_t K, size_t N) {
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
    matmul_kernel<<<gridDim, blockDim>>>(d_in1, d_in2, d_out, M, K, N);
}

/* Transpose
 * @param [in1]  in: [M, N]
 * @param [out] out: [N, M]
 */
// void transpose(Tensor *in, Tensor *out) {
//   size_t M = in->shape[0];
//   size_t N = in->shape[1];

//   for (size_t i = 0; i < M; i++) {
//     for (size_t j = 0; j < N; j++) { out->buf[j * M + i] = in->buf[i * N + j]; }
//   }
// }

__global__ void transpose_kernel(float *in, float *out, size_t M, size_t N) {
  size_t row = blockIdx.y * blockDim.y + threadIdx.y; 
  size_t col = blockIdx.x * blockDim.x + threadIdx.x; 
  
  if (row < M && col < N) {
    out[col * M + row] = in[row * N + col];
  }
}

void transpose(Tensor *in, Tensor *out) {
  size_t M = in->shape[0];
  size_t N = in->shape[1]; 

  float *d_in;
  float *d_out;

  hipMalloc(&d_in, M * N * sizeof(float));
  hipMalloc(&d_out, N * M * sizeof(float));

  hipMemcpy(d_in, in->buf, M * N * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(16, 16);
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim. y - 1) / blockDim.y);
  transpose_kernel<<<gridDim, blockDim>>>(d_in, d_out, M, N); 

  hipMemcpy(out->buf, d_out, M * N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);
}

void transpose(float *d_in, float *d_out, size_t M, size_t N) {
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);
    transpose_kernel<<<gridDim, blockDim>>>(d_in, d_out, M, N); 
}

/* Scaling
 * @param [in1 & out] inout: [N]
 * @param [in2]       scale: [1]
 * 'N' is the number of elements in the tensor.
 */
// void scaling(Tensor *inout, float scale) {
//   size_t N = inout->num_elem();

//   for (size_t i = 0; i < N; i++) { inout->buf[i] *= scale; }
// }

__global__ void scaling_kernel(float *inout, float scale, size_t N) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x; 

  if (idx < N) {
    inout[idx] *= scale; 
  }
}

void scaling(Tensor *inout, float scale) {
  size_t N = inout->num_elem();

  float *d_inout; 

  hipMalloc(&d_inout, N * sizeof(float));

  hipMemcpy(d_inout, inout->buf, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(256);
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
  scaling_kernel<<<gridDim, blockDim>>>(d_inout, scale, N); 

  hipMemcpy(inout->buf, d_inout, N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_inout); 
}

void scaling(float *d_inout, float scale, size_t N) {
    dim3 blockDim(256);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
    scaling_kernel<<<gridDim, blockDim>>>(d_inout, scale, N); 
}

/* Generate mask
 * @param [in & out] inout: [s, s]
 * 's' is the number of tokens in the prompt.
 */
// void generate_mask(Tensor *inout) {
//   size_t s = inout->shape[0];

//   for (size_t i = 0; i < s; i++) {
//     for (size_t j = 0; j < s; j++) {
//       if (i >= j) {
//         inout->buf[i * s + j] = 0;
//       } else {
//         inout->buf[i * s + j] = -1e10;
//       }
//     }
//   }
// }

__global__ void generate_mask_kernel(float *inout, size_t s) {
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < s && col < s) {
    if (row >= col) {
      inout[row * s + col] = 0;
    } else {
      inout[row * s + col] = -1e10;
    }
  }
}

void generate_mask(Tensor *inout) {
  size_t s = inout->shape[0];

  float *d_inout;

  hipMalloc(&d_inout, s * s * sizeof(float));

  hipMemcpy(d_inout, inout->buf, s * s * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(16, 16);
  dim3 gridDim((s + blockDim.x - 1) / blockDim.x, (s + blockDim.y - 1) / blockDim.y);
  generate_mask_kernel<<<gridDim, blockDim>>>(d_inout, s);

  hipMemcpy(inout->buf, d_inout, s * s * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_inout);
}

void generate_mask(float *d_inout, size_t s) {
    dim3 blockDim(16, 16);
    dim3 gridDim((s + blockDim.x - 1) / blockDim.x, (s + blockDim.y - 1) / blockDim.y);
    generate_mask_kernel<<<gridDim, blockDim>>>(d_inout, s);
}

/* Copy
 * @param [in1]  in: [N]
 * @param [out] out: [N]
 * 'N' is the number of elements in the tensor.
 */
// void copy(Tensor *in, Tensor *out) {
//   size_t N = in->num_elem();

//   for (size_t i = 0; i < N; i++) { out->buf[i] = in->buf[i]; }
// }

__global__ void copy_kernel(float *in, float *out, size_t N) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x; 

  if (idx < N) {
    out[idx] = in[idx];
  }
}

void copy(Tensor *in, Tensor *out) {
  size_t N = in->num_elem(); 

  float *d_in;
  float *d_out; 

  hipMalloc(&d_in, N * sizeof(float));
  hipMalloc(&d_out, N * sizeof(float)); 

  hipMemcpy(d_in, in->buf, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(256);
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
  copy_kernel<<<gridDim, blockDim>>>(d_in, d_out, N);

  hipMemcpy(out->buf, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out); 
}

void copy(float *d_in, float *d_out, size_t N) {
    dim3 blockDim(256);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
    copy_kernel<<<gridDim, blockDim>>>(d_in, d_out, N);
}

/* Add
 * @param [in1 & out] inout: [N]
 * @param [in2]           x: [N]
 * 'N' is the number of elements in the tensor.
 */
// void add(Tensor *inout, Tensor *x) {
//   size_t N = inout->num_elem();

//   for (size_t i = 0; i < N; i++) { inout->buf[i] += x->buf[i]; }
// }

/* Add GPU kernel
 * @param [in1 & out] inout: [N]
 * @param [in2]           x: [N]
 * 'N' is the number of elements in the tensor.
 */
__global__ void add_kernel(float *inout, float *x, size_t N) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) { inout[idx] += x[idx]; }
}

/* Add using CUDA GPU
 * @param [in1 & out] inout: [N]
 * @param [in2]           x: [N]
 * 'N' is the number of elements in the tensor.
 */
void add(Tensor *inout, Tensor *x) {
  size_t N = inout->num_elem();

  float *d_inout;
  float *d_x;

  hipMalloc(&d_inout, N * sizeof(float));
  hipMalloc(&d_x, N * sizeof(float));

  hipMemcpy(d_inout, inout->buf, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_x, x->buf, N * sizeof(float), hipMemcpyHostToDevice);

  add_kernel<<<(N + 255) / 256, 256>>>(d_inout, d_x, N);

  hipMemcpy(inout->buf, d_inout, N * sizeof(float), hipMemcpyDeviceToHost);
}

void add(float *d_inout, float *d_x, size_t N) {
    dim3 blockDim(256);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
    add_kernel<<<gridDim, blockDim>>>(d_inout, d_x, N);
}

/* Split into QKV
 * @param [in1]  in: [s, H]
 * @param [out] out: [3, s, H/3]
 */
// void split_qkv(Tensor *in, Tensor *out) {
//   size_t s = in->shape[0];
//   size_t H = in->shape[1];

//   for (size_t i = 0; i < 3; i++) {
//     for (size_t j = 0; j < s; j++) {
//       for (size_t k = 0; k < H / 3; k++) {
//         out->buf[i * s * (H / 3) + j * (H / 3) + k] =
//             in->buf[i * (H / 3) + j * 3 * (H / 3) + k];
//       }
//     }
//   }
// }

__global__ void split_qkv_kernel(float *in, float *out, size_t s, size_t H, size_t N) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    size_t i = idx / (s * (H / 3));
    size_t j = (idx % (s * (H / 3))) / (H / 3);
    size_t k = idx % (H / 3);

    out[idx] = in[i * (H / 3) + j * H + k];
  }
}

void split_qkv(Tensor *in, Tensor *out) {
  size_t s = in->shape[0];
  size_t H = in->shape[1];
  size_t N = 3 * s * (H / 3);

  float *d_in;
  float *d_out;

  hipMalloc(&d_in, s * H * sizeof(float));
  hipMalloc(&d_out, s * H * sizeof(float));

  hipMemcpy(d_in, in->buf, s * H * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(256);
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
  split_qkv_kernel<<<gridDim, blockDim>>>(d_in, d_out, s, H, N);

  hipMemcpy(out->buf, d_out, s * H * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);
}

void split_qkv(float *d_in, float *d_out, size_t s, size_t H) {
    dim3 blockDim(256);
    dim3 gridDim((s * H + blockDim.x - 1) / blockDim.x);
    split_qkv_kernel<<<gridDim, blockDim>>>(d_in, d_out, s, H, s * H);
}

/* Split into heads
 * @param [in1]  in: [3, s, H]
 * @param [out] out: [3, n_head, s, H/n_head]
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 * 'n_head' is the number of heads.
 */
// void split_head(Tensor *in, size_t n_head, Tensor *out) {
//   size_t s = in->shape[1];
//   size_t H = in->shape[2];

//   for (size_t i = 0; i < 3; i++) {
//     for (size_t j = 0; j < n_head; j++) {
//       for (size_t k = 0; k < s; k++) {
//         for (size_t l = 0; l < H / n_head; l++) {
//           out->buf[i * n_head * s * H / n_head + j * s * H / n_head +
//                    k * H / n_head + l] =
//               in->buf[i * s * H + k * H + j * H / n_head + l];
//         }
//       }
//     }
//   }
// }

__global__ void split_head_kernel(float *in, float *out, size_t n_head, size_t s, size_t H, size_t N) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    size_t i = idx / (n_head * s * (H / n_head));
    size_t j = (idx % (n_head * s * (H / n_head))) / (s * (H / n_head));
    size_t k = (idx % (s * (H / n_head))) / (H / n_head);
    size_t l = idx % (H / n_head);

    out[idx] = in[i * s * H + k * H + j * (H / n_head) + l];
  }
}

void split_head(Tensor *in, size_t n_head, Tensor *out) {
  size_t s = in->shape[1];
  size_t H = in->shape[2];
  size_t N = 3 * n_head * s * (H / n_head);

  float *d_in;
  float *d_out;

  hipMalloc(&d_in, 3 * s * H * sizeof(float));
  hipMalloc(&d_out, 3 * n_head * s * (H / n_head) * sizeof(float));

  hipMemcpy(d_in, in->buf, 3 * s * H * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(256);
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
  split_head_kernel<<<gridDim, blockDim>>>(d_in, d_out, n_head, s, H, N);

  hipMemcpy(out->buf, d_out, 3 * n_head * s * (H / n_head) * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);
}

void split_head(float *d_in, float *d_out, size_t n_head, size_t s, size_t H) {
    dim3 blockDim(256);
    dim3 gridDim((3 * s * H + blockDim.x - 1) / blockDim.x);
    split_head_kernel<<<gridDim, blockDim>>>(d_in, d_out, n_head, s, H, 3 * s * H);
}

/* Extract Q, K, V from QKV head
 * @param [in1]       in: [3, n_head, s, H_]
 * @param [in2] head_idx: [1]
 * @param [in3]   n_head: [1]
 * @param [out]        q: [s, H_]
 * @param [out]        k: [s, H_]
 * @param [out]        v: [s, H_]
 * 's' is the number of tokens in the prompt.
 * 'H_' is the hidden dimension/n_head.
 * 'n_head' is the number of heads.
 */
// void extract_qkv(Tensor *in, size_t head_idx, size_t n_head, Tensor *q,
//                  Tensor *k, Tensor *v) {
//   size_t s = in->shape[2];
//   size_t H_ = in->shape[3];  // = HIDDEN_DIM/NUM_HEAD

//   for (size_t i = 0; i < s; i++) {
//     for (size_t j = 0; j < H_; j++) {
//       q->buf[i * H_ + j] =
//           in->buf[0 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
//       k->buf[i * H_ + j] =
//           in->buf[1 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
//       v->buf[i * H_ + j] =
//           in->buf[2 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
//     }
//   }
// }

__global__ void extract_qkv_kernel(float *in, size_t head_idx, size_t n_head, float *q, float *k, float *v, size_t s, size_t H_, size_t N) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    size_t i = idx / H_;
    size_t j = idx % H_;

    q[i * H_ + j] = in[0 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
    k[i * H_ + j] = in[1 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
    v[i * H_ + j] = in[2 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
  }
}

void extract_qkv(Tensor *in, size_t head_idx, size_t n_head, Tensor *q, Tensor *k, Tensor *v) {
  size_t s = in->shape[2];
  size_t H_ = in->shape[3];
  size_t N = s * H_;

  float *d_in;
  float *d_q;
  float *d_k;
  float *d_v;

  hipMalloc(&d_in, 3 * n_head * s * H_ * sizeof(float));
  hipMalloc(&d_q, s * H_ * sizeof(float));
  hipMalloc(&d_k, s * H_ * sizeof(float));
  hipMalloc(&d_v, s * H_ * sizeof(float));

  hipMemcpy(d_in, in->buf, 3 * n_head * s * H_ * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(256);
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
  extract_qkv_kernel<<<gridDim, blockDim>>>(d_in, head_idx, n_head, d_q, d_k, d_v, s, H_, N);

  hipMemcpy(q->buf, d_q, s * H_ * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(k->buf, d_k, s * H_ * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(v->buf, d_v, s * H_ * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_q);
  hipFree(d_k);
  hipFree(d_v);
}

void extract_qkv(float *d_in, float *d_q, float *d_k, float *d_v, size_t head_idx, size_t n_head, size_t s, size_t H_) {
    dim3 blockDim(256);
    dim3 gridDim((s * H_ + blockDim.x - 1) / blockDim.x);
    extract_qkv_kernel<<<gridDim, blockDim>>>(d_in, head_idx, n_head, d_q, d_k, d_v, s, H_, s * H_);
}

/* Merge each heads
 * @param [in1]       in: [s, H_]
 * @param [in2] head_idx: [1]
 * @param [in3]   n_head: [1]
 * @param [out]      out: [n_head, s, H_]
 * 's' is the number of tokens in the prompt.
 * 'H_' is the hidden dimension/n_head.
 * 'n_head' is the number of heads.
 */
// void merge_head(Tensor *in, size_t head_idx, size_t n_head, Tensor *out) {
//   size_t s = in->shape[0];
//   size_t H_ = in->shape[1];  // = HIDDEN_DIM/NUM_HEAD

//   for (size_t i = 0; i < s; i++) {
//     for (size_t j = 0; j < H_; j++) {
//       out->buf[head_idx * s * H_ + i * H_ + j] = in->buf[i * H_ + j];
//     }
//   }
// }

__global__ void merge_head_kernel(float *in, size_t head_idx, size_t s, size_t H_, float *out) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < s * H_) {
    size_t i = idx / H_;
    size_t j = idx % H_;
    out[head_idx * s * H_ + i * H_ + j] = in[idx];
  }
}

void merge_head(Tensor *in, size_t head_idx, size_t n_head, Tensor *out) {
  size_t s = in->shape[0];
  size_t H_ = in->shape[1];

  float *d_in;
  float *d_out;

  hipMalloc(&d_in, s * H_ * sizeof(float));
  hipMalloc(&d_out, n_head * s * H_ * sizeof(float));  

  hipMemcpy(d_in, in->buf, s * H_ * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_out, out->buf, n_head * s * H_ * sizeof(float), hipMemcpyHostToDevice);  

  dim3 blockDim(256);
  dim3 gridDim((s * H_ + blockDim.x - 1) / blockDim.x);
  merge_head_kernel<<<gridDim, blockDim>>>(d_in, head_idx, s, H_, d_out);

  hipMemcpy(out->buf, d_out, n_head * s * H_ * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);
}

void merge_head(float *d_in, float *d_out, size_t head_idx, size_t s, size_t H_) {
    dim3 blockDim(256);
    dim3 gridDim((s * H_ + blockDim.x - 1) / blockDim.x);
    merge_head_kernel<<<gridDim, blockDim>>>(d_in, head_idx, s, H_, d_out);
}

/* Concatenate each heads
 * @param [in1]     in: [n_head, s, H_]
 * @param [out]    out: [s, H_*n_head]
 * 'n_head' is the number of heads.
 * 's' is the number of tokens in the prompt.
 * 'H_' is the hidden dimension/n_head.
 */
// void concat_head(Tensor *in, Tensor *out) {
//   size_t n_head = in->shape[0];
//   size_t s = in->shape[1];
//   size_t H_ = in->shape[2];  // = HIDDEN_DIM/NUM_HEAD

//   for (size_t i = 0; i < s; i++) {
//     for (size_t j = 0; j < n_head; j++) {
//       for (size_t k = 0; k < H_; k++) {
//         out->buf[i * n_head * H_ + j * H_ + k] =
//             in->buf[j * s * H_ + i * H_ + k];
//       }
//     }
//   }
// }

__global__ void concat_head_kernel(float *in, float *out, size_t n_head, size_t s, size_t H_) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x; 

  if (idx < s * H_ * n_head) {
    size_t i = (idx % (s * H_)) / H_; 
    size_t j = idx / (s * H_); 
    size_t k = idx % H_; 
    out[i * n_head * H_ + j * H_ + k] = in[j * s * H_ + i * H_ + k];
  }
}

void concat_head(Tensor *in, Tensor *out) {
  size_t n_head = in->shape[0];
  size_t s = in->shape[1]; 
  size_t H_ = in->shape[2]; 

  float *d_in; 
  float *d_out; 

  hipMalloc(&d_in, n_head * s * H_ * sizeof(float));
  hipMalloc(&d_out, s * H_ * n_head * sizeof(float));

  hipMemcpy(d_in, in->buf, n_head * s * H_ * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockDim(256);
  dim3 gridDim((n_head * s * H_ + blockDim.x - 1) / blockDim.x);
  concat_head_kernel<<<gridDim, blockDim>>>(d_in, d_out, n_head, s, H_);

  hipMemcpy(out->buf, d_out, s * n_head * H_ * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);
}

void concat_head(float *d_in, float *d_out, size_t n_head, size_t s, size_t H_) {
    dim3 blockDim(256);
    dim3 gridDim((n_head * s * H_ + blockDim.x - 1) / blockDim.x);
    concat_head_kernel<<<gridDim, blockDim>>>(d_in, d_out, n_head, s, H_);
}

/* Greedy Max Sampling
 * @param  [in1]  in: [s, V]
 * @return [ret] out: [1]
 * 's' is the number of tokens in the prompt.
 * 'V' is the number of vocabulary.
 */
int top1_sampling(Tensor *in) {
  size_t s = in->shape[0];
  size_t V = in->shape[1];

  int out = 0;
  float max = -INFINITY;
  for (size_t i = 0; i < V; i++) {
    if (in->buf[(s - 1) * V + i] > max) {
      max = in->buf[(s - 1) * V + i];
      out = i;
    }
  }

  return out;
}